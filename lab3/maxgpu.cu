#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>

__global__ void getmaxcu(unsigned int num[], unsigned int size, unsigned int gap)
{
  unsigned int i=gap,                                                      //loop variables
               start = (threadIdx.x)*gap;
  if(start%2!=0 || size<=0)
    return;
  else{
    //bottom-up tree search
    while(start+i<size && start%(2*i)==0){
      if(num[start]<num[start+i])
        num[start]=num[start+i];
      i*=2;
      __syncthreads();
    }
  }
}

int main(int argc, char *argv[])
{
    if(argc !=2)
    {
       printf("usage: maxgpu num\n");
       printf("num = size of the array\n");
       exit(1);
    }

    hipError_t  err;                                                   // error var
    unsigned int size = 0, backup_size,                                 // The size of array
                 thread_num,
                 i;                                                     // loop index
    struct hipDeviceProp_t prop;                                         // specs
    unsigned int *numbers, *cudanumbers,                                // pointers to array
                 *max=(unsigned int *)malloc(sizeof(unsigned int));     // pointers to max number
   
    size = atol(argv[1]);
    backup_size=size;
    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));

    if(!numbers)
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }    
    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
      numbers[i] = rand() % size;

    hipGetDeviceProperties(&prop, 0);
    hipMalloc((void**)&cudanumbers, size * sizeof(unsigned int));
    hipMemcpy(cudanumbers, numbers, size * sizeof(unsigned int), hipMemcpyHostToDevice);
    thread_num=ceil(prop.maxThreadsPerBlock/32)*32;
    unsigned int offset=0;
    i=0;
    //search 1024 elements at once; search remaining elements
    //structure: |----1024----| |----1024----| ... |size%1024| left to right
    while(offset<backup_size){
      if(backup_size-offset>=thread_num){
        getmaxcu<<<1, thread_num>>>((cudanumbers+offset), thread_num, 1);
      }
      else{
        getmaxcu<<<1, backup_size-offset>>>((cudanumbers+offset), backup_size-offset, 1);
      }
      getmaxcu<<<1, 1>>>((cudanumbers), backup_size, offset);
      offset+=thread_num;
    }
    hipMemcpy(max, cudanumbers, sizeof(unsigned int), hipMemcpyDeviceToHost);
    err=hipGetLastError();
    if(err!=hipSuccess){
      printf("CUDA error: %s\n", hipGetErrorString(err));
      exit(-1);
    }
    hipDeviceSynchronize();
    hipFree(cudanumbers);

    printf("CUDA returns: %d\n", *max);
    //printf("Sequential returns: %u\n", getmax(numbers, backup_size));

    free(numbers);
    free(max);

    exit(0);
}

/*
unsigned int getmax(unsigned int num[], unsigned int size)
{
  unsigned int i, j;
  unsigned int max = num[0];

  for(i = 1; i < size; i++)
    if(num[i] > max){
        max = num[i];
        j=i;
    }

  //printf("Sequential found max at %d\n", j);
  return max;
}

__global__ void printArray(unsigned int arr[], unsigned int length, unsigned int jump){
  for(int i = 0; i+jump<length; i++)
    printf("%d: %d\n", i+jump, arr[i+jump]);
}

void search(unsigned int arr[], unsigned int size, unsigned int target){
  printf("CUDA found max at");
  for(int i = 1; i < size; i++)
    if(arr[i] == target)
       printf(" %d ", i);
  printf("\n");
}
*/